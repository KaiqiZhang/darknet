#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include "hipfft/hipfft.h"

extern "C" {
#include "convolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    binary[i] = (x[i] >= 0) ? 1 : -1;
}

void binarize_gpu(float *x, int n, float *binary)
{
    binarize_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, binary);
    check_error(hipPeekAtLastError());
}

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < n; ++i){
        mean += fabsf(input[i*size + s]);
    }
    mean = mean / n;
    for(i = 0; i < n; ++i){
        binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
    }
}

void binarize_input_gpu(float *input, int n, int size, float *binary)
{
    binarize_input_kernel<<<cuda_gridsize(size), BLOCK>>>(input, n, size, binary);
    check_error(hipPeekAtLastError());
}


__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < size; ++i){
        mean += fabsf(weights[f*size + i]);
    }
    mean = mean / size;
    for(i = 0; i < size; ++i){
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
        //binary[f*size + i] = weights[f*size + i];
    }
}

void binarize_weights_gpu(float *weights, int n, int size, float *binary)
{
    binarize_weights_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, size, binary);
    check_error(hipPeekAtLastError());
}

__global__ void complex_mul_kernel(int n, hipfftComplex *b, hipfftComplex *zweights, hipfftComplex *output,
                                  int block_row, int block_col, int block_size_half, int out_h, int out_w)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if (i >= n) return;

  // calculate one block element-wise multiplication
  int col = i % (out_h * out_w * block_row);
  int row = i / block_col % (out_h * out_w);
  int pixel = i / block_row / block_col;
  hipfftComplex *im_ptr = b + pixel * block_col * block_size_half + col * block_size_half;
  hipfftComplex *weight_ptr = zweights + row * block_col * block_size_half + col * block_size_half;
  hipfftComplex *output_ptr = output + pixel * block_row * block_col * block_size_half + row * block_col * block_size_half + col * block_size_half;

  for (int k = 0; k < block_size_half; k++)
  {
    //output_ptr->x = im_ptr->x * weight_ptr->x - im_ptr->y * weight_ptr->y;
    //output_ptr->y = im_ptr->x * weight_ptr->y + im_ptr->y * weight_ptr->x;
    output_ptr->x = im_ptr->x * weight_ptr->x - im_ptr->y * weight_ptr->y;
    output_ptr->y = im_ptr->x * weight_ptr->y + im_ptr->y * weight_ptr->x;

    im_ptr++;
    weight_ptr++;
    output_ptr++;
  }
}

void forward_convolutional_layer_gpu(convolutional_layer l, network net)
{
    fill_gpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    if(l.binary){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c/l.groups*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
    }

    if(l.xnor){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c/l.groups*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
        binarize_gpu(net.input_gpu, l.c*l.h*l.w*l.batch, l.binary_input_gpu);
        net.input_gpu = l.binary_input_gpu;
    }

#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionForward(cudnn_handle(),
                &one,
                l.srcTensorDesc,
                net.input_gpu,
                l.weightDesc,
                l.weights_gpu,
                l.convDesc,
                l.fw_algo,
                net.workspace,
                l.workspace_size,
                &one,
                l.dstTensorDesc,
                l.output_gpu);

#else
    int i, j;
    int m = l.n/l.groups;
    int k = l.size*l.size*l.c/l.groups;
    int n = l.out_w*l.out_h;
    for(i = 0; i < l.batch; ++i){
        for(j = 0; j < l.groups; ++j){
            float *a = l.weights_gpu + j*l.nweights/l.groups;
            float *b = net.workspace;
            float *c = l.output_gpu + (i*l.groups + j)*n*m;
            float *im = net.input_gpu + (i*l.groups + j)*l.c/l.groups*l.h*l.w;

            if (l.size == 1){
                b = im;
            } else {
                im2col_gpu(im, l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, b);
            }
            //im2col_transpose_gpu();

            //gemm_gpu(0,0,m,n,k,1,a,k,b,n,1,c,n);
            // group not considered !!!

            // transpose b?
            hipfftExecR2C(l.fft_b_plan, (hipfftReal *)b, l.fft_b_gpu);
            check_error(hipPeekAtLastError());

            int n = l.out_h*l.out_w*l.block_col*l.block_row;
            complex_mul_kernel<<<cuda_gridsize(n), BLOCK>>>(n, l.fft_b_gpu, l.fft_zweights_gpu, l.fft_output_mul_gpu,
                                                       l.block_row, l.block_col, l.block_size_half, l.out_h, l.out_w);
            check_error(hipPeekAtLastError());

            hipfftExecC2R(l.ifft_plan, l.fft_output_gather_gpu, (hipfftReal *)c);
            check_error(hipPeekAtLastError());

            // transpose output
            const float one = 1;
	          const float zero = 0;
            hipblasSgeam(0, HIPBLAS_OP_T, HIPBLAS_OP_T, l.block_col*l.block_size, n, &one, c, n, &zero, c, n, c, l.block_col*l.block_size);
            check_error(hipPeekAtLastError());

            //invert<<<>>>(l.fft_output_gpu, l.output_gpu, c, m*n);  // add & transpose & complex2real
        }
    }
#endif

    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, net);
    } else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

    activate_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation);
    //if(l.dot > 0) dot_error_gpu(l);
    if(l.binary || l.xnor) swap_binary(&l);
}

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -(size/2.f);
    int h_offset = -(size/2.f);

    int out_index = j + w*(i + h*(k + c*b));
    int l, m;
    for(l = 0; l < size; ++l){
        for(m = 0; m < size; ++m){
            int cur_h = h_offset + i + l;
            int cur_w = w_offset + j + m;
            int index = cur_w + w*(cur_h + h*(k + b*c));
            int valid = (cur_h >= 0 && cur_h < h &&
                    cur_w >= 0 && cur_w < w);
            delta[out_index] += valid ? rate*(x[index] - x[out_index]) : 0;
        }
    }
}

extern "C" void smooth_layer(layer l, int size, float rate)
{
    int h = l.out_h;
    int w = l.out_w;
    int c = l.out_c;

    size_t n = h*w*c*l.batch;

    smooth_kernel<<<cuda_gridsize(n), BLOCK>>>(l.output_gpu, n, l.w, l.h, l.c, size, rate, l.delta_gpu);
    check_error(hipPeekAtLastError());
}

void backward_convolutional_layer_gpu(convolutional_layer l, network net)
{
    if(l.smooth){
        smooth_layer(l, 5, l.smooth);
    }
    //constrain_gpu(l.outputs*l.batch, 1, l.delta_gpu, 1);
    gradient_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);


    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, net);
    } else {
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
    float *original_input = net.input_gpu;

    if(l.xnor) net.input_gpu = l.binary_input_gpu;
#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionBackwardFilter(cudnn_handle(),
            &one,
            l.srcTensorDesc,
            net.input_gpu,
            l.ddstTensorDesc,
            l.delta_gpu,
            l.convDesc,
            l.bf_algo,
            net.workspace,
            l.workspace_size,
            &one,
            l.dweightDesc,
            l.weight_updates_gpu);

    if(net.delta_gpu){
        if(l.binary || l.xnor) swap_binary(&l);
        hipdnnConvolutionBackwardData(cudnn_handle(),
                &one,
                l.weightDesc,
                l.weights_gpu,
                l.ddstTensorDesc,
                l.delta_gpu,
                l.convDesc,
                l.bd_algo,
                net.workspace,
                l.workspace_size,
                &one,
                l.dsrcTensorDesc,
                net.delta_gpu);
        if(l.binary || l.xnor) swap_binary(&l);
        if(l.xnor) gradient_array_gpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, net.delta_gpu);
    }

#else
    int m = l.n/l.groups;
    int n = l.size*l.size*l.c/l.groups;
    int k = l.out_w*l.out_h;

    int i, j;
    for(i = 0; i < l.batch; ++i){
        for(j = 0; j < l.groups; ++j){
            float *a = l.delta_gpu + (i*l.groups + j)*m*k;
            float *b = net.workspace;
            float *c = l.weight_updates_gpu + j*l.nweights/l.groups;

            float *im  = net.input_gpu+(i*l.groups + j)*l.c/l.groups*l.h*l.w;
            float *imd = net.delta_gpu+(i*l.groups + j)*l.c/l.groups*l.h*l.w;

            im2col_gpu(im, l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, b);
            gemm_gpu(0,1,m,n,k,1,a,k,b,k,1,c,n);

            if (net.delta_gpu) {
                if (l.binary || l.xnor) swap_binary(&l);
                a = l.weights_gpu + j*l.nweights/l.groups;
                b = l.delta_gpu + (i*l.groups + j)*m*k;
                c = net.workspace;
                if (l.size == 1) {
                    c = imd;
                }

                gemm_gpu(1,0,n,k,m,1,a,n,b,k,0,c,k);

                if (l.size != 1) {
                    col2im_gpu(net.workspace, l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, imd);
                }
                if(l.binary || l.xnor) {
                    swap_binary(&l);
                }
            }
            if(l.xnor) gradient_array_gpu(original_input + i*l.c*l.h*l.w, l.c*l.h*l.w, HARDTAN, net.delta_gpu + i*l.c*l.h*l.w);
        }
    }
#endif
}

void pull_convolutional_layer(layer l)
{
    cuda_pull_array(l.weights_gpu, l.weights, l.nweights);
    cuda_pull_array(l.biases_gpu, l.biases, l.n);
    cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
    cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_pull_array(l.scales_gpu, l.scales, l.n);
        cuda_pull_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_pull_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void push_convolutional_layer(layer l)
{
    cuda_push_array(l.weights_gpu, l.weights, l.nweights);
    cuda_push_array(l.biases_gpu, l.biases, l.n);
    cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
    cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_push_array(l.scales_gpu, l.scales, l.n);
        cuda_push_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_push_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void update_convolutional_layer_gpu(layer l, update_args a)
{
    float learning_rate = a.learning_rate*l.learning_rate_scale;
    float momentum = a.momentum;
    float decay = a.decay;
    int batch = a.batch;

    if(a.adam){
        adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.nweights, batch, a.t);
        adam_update_gpu(l.biases_gpu, l.bias_updates_gpu, l.bias_m_gpu, l.bias_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        if(l.scales_gpu){
            adam_update_gpu(l.scales_gpu, l.scale_updates_gpu, l.scale_m_gpu, l.scale_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        }
    }else{
        axpy_gpu(l.nweights, -decay*batch, l.weights_gpu, 1, l.weight_updates_gpu, 1);
        axpy_gpu(l.nweights, learning_rate/batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);
        scal_gpu(l.nweights, momentum, l.weight_updates_gpu, 1);

        axpy_gpu(l.n, learning_rate/batch, l.bias_updates_gpu, 1, l.biases_gpu, 1);
        scal_gpu(l.n, momentum, l.bias_updates_gpu, 1);

        if(l.scales_gpu){
            axpy_gpu(l.n, learning_rate/batch, l.scale_updates_gpu, 1, l.scales_gpu, 1);
            scal_gpu(l.n, momentum, l.scale_updates_gpu, 1);
        }
    }
    if(l.clip){
        constrain_gpu(l.nweights, l.clip, l.weights_gpu, 1);
    }
}
